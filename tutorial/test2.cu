
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

// 函数用于将字符串中的坐标提取到数组中
std::vector<double> extractCoordinates(const std::string& wkt) {
    std::vector<double> coordinates;
    size_t start = wkt.find("(") + 1;  // 寻找坐标开始的位置
    size_t end = wkt.find(")");        // 寻找坐标结束的位置

    if (start != std::string::npos && end != std::string::npos) {
        std::string coordinatesStr = wkt.substr(start, end - start);  // 提取坐标部分
        std::istringstream iss(coordinatesStr);
        double coordinate;
        while (iss >> coordinate) {
            coordinates.push_back(coordinate);
        }
    }

    return coordinates;
}

int main() {
    // 文件路径
    std::string filename = "bigpolygon.wkt";

    // 创建文件输入流对象
    std::ifstream inputFile(filename);

    // 检查文件是否成功打开
    if (!inputFile.is_open()) {
        std::cerr << "无法打开文件" << std::endl;
        return 1;
    }

    std::vector<std::vector<double>> polygonCoordinates;

    // 逐行读取文件内容
    std::string line;
    while (std::getline(inputFile, line)) {
        // 提取坐标并存储到数组中
        std::vector<double> coordinates = extractCoordinates(line);
        if (!coordinates.empty()) {
            polygonCoordinates.push_back(coordinates);
        }
    }

    // 关闭文件
    inputFile.close();

    // 输出提取的坐标
    for (const auto& coordinates : polygonCoordinates) {
        for (double coordinate : coordinates) {
            std::cout << coordinate << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}
