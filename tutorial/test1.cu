
#include <hip/hip_runtime.h>
#include <cstdio>
using namespace std;

__global__ void hello_from_gpu() {
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int bz = blockIdx.z;
}

int main() {
    const dim3 block_size(2, 4);
    // grid的维度，说明了block的数量
    // block的维度，说明了thread的数量
    hello_from_gpu<<<2, block_size>>>();
    hipDeviceSynchronize();
}